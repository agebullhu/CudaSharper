#pragma once
#include "device_info.h"

namespace csl {

	int32_t marshal_cuda_error(hipError_t error) {
		return (int)error;
	}

	device_info::device_info(int32_t device_id) {
		this->device_id = device_id;
	}

	hipError_t device_info::get_device_properties(int32_t device_id, hipDeviceProp_t *prop) const {
		hipGetDeviceProperties(prop, device_id);

		return hipSuccess;
	}

	int32_t device_info::get_cuda_device_count() const {
		int32_t num = 0;
		this->get_cuda_device_count(num);
		return num;
	}

	hipError_t device_info::get_cuda_device_count(int32_t& result) const {
		int num = 0;
		hipError_t errorCode = hipGetDeviceCount(&num);
		if (errorCode != hipSuccess) return errorCode;

		result = num;

		return hipSuccess;
	}

	hipError_t device_info::get_cuda_device_name(int32_t device_id, char* device_name_ptr) const {
		hipDeviceProp_t prop;
		// The following isn't working correctly. Eventually, it should be used, because then we can save time calling hipGetDeviceProperties
		hipError_t errorCode = this->get_device_properties(device_id, &prop);
		if (errorCode != hipSuccess) return errorCode;

		// Length of hipDeviceProp_t::name, according to current NVIDIA documentation: http://docs.nvidia.com/cuda/cuda-runtime-api/structcudaDeviceProp.html#structcudaDeviceProp_11e26f1c6bd42f4821b7ef1a4bd3bd25c
		strncpy(device_name_ptr, prop.name, 256);

		return hipSuccess;
	}

	hipError_t device_info::reset_cuda_device() const {
		hipError_t errorCode;
		for (int i = 0; i < this->get_cuda_device_count(); i++) {
			errorCode = hipSetDevice(i);
			if (errorCode != hipSuccess) return errorCode;
			errorCode = hipDeviceReset();
			if (errorCode != hipSuccess) return errorCode;
		}
		return hipSuccess;
	}

	int32_t device_info::get_device_id() const {
		return this->device_id;
	}

}

