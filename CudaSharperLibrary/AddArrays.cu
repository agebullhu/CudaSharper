#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "hip/device_functions.h"

/*
 Functions for adding two arrays together. Requires CUDA 8.0.
*/

__global__ void addArraysKernel(int *result, int *a, int *b, unsigned int array_count) {
	int xid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (xid < array_count) {
		result[xid] = a[xid] + b[xid];
	}
}

__global__ void addArraysKernel(float *result, float *a, float *b, unsigned int array_count) {
	int xid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (xid < array_count) {
		result[xid] = a[xid] + b[xid];
	}
}

__global__ void addArraysKernel(double *result, double *a, double *b, unsigned int array_count) {
	int xid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (xid < array_count) {
		result[xid] = a[xid] + b[xid];
	}
}

__global__ void addArraysKernel(long *result, long *a, long *b, unsigned int array_count) {
	int xid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (xid < array_count) {
		result[xid] = a[xid] + b[xid];
	}
}

template<typename T> void addArrays(int device_id, T *result, T *array1, T *array2, const int full_idx) {
	hipError_t gpu_device = hipSetDevice(device_id);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device_id);

	const int threadsPerBlock = prop.maxThreadsPerBlock;
	const int blocks = (full_idx / threadsPerBlock) + 1;

	T *d_a, *d_b, *dResult;

	hipMalloc(&d_a, sizeof(T) * full_idx);
	hipMalloc(&d_b, sizeof(T) * full_idx);
	hipMalloc(&dResult, sizeof(T) * full_idx);

	hipMemcpy(d_a, array1, sizeof(T) * full_idx, hipMemcpyHostToDevice);
	hipMemcpy(d_b, array2, sizeof(T) * full_idx, hipMemcpyHostToDevice);
	hipMemcpy(dResult, result, sizeof(T) * full_idx, hipMemcpyHostToDevice);

	addArraysKernel << <blocks, threadsPerBlock >> > (dResult, d_a, d_b, full_idx);

	hipMemcpy(result, dResult, sizeof(T) * full_idx, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(dResult);
}

extern "C" __declspec(dllexport) void AddIntArrays(int device_id, int *result, int *array1, int *array2, const int full_idx) {
	addArrays<int>(device_id, result, array1, array2, full_idx);
}

extern "C" __declspec(dllexport) void AddFloatArrays(int device_id, float *result, float *array1, float *array2, const int full_idx) {
	addArrays<float>(device_id, result, array1, array2, full_idx);
}

extern "C" __declspec(dllexport) void AddLongArrays(int device_id, long *result, long *array1, long *array2, const int full_idx) {
	addArrays<long>(device_id, result, array1, array2, full_idx);
}

extern "C" __declspec(dllexport) void AddDoubleArrays(int device_id, double *result, double *array1, double *array2, const int full_idx) {
	addArrays<double>(device_id, result, array1, array2, full_idx);
}

