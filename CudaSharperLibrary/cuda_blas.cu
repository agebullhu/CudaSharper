#include "cuda_blas.h"

hipblasOperation_t cuda_blas_determine_op(int32_t op) {
	hipblasOperation_t trans_op = HIPBLAS_OP_N;
	switch (op) {
		case 0: trans_op = HIPBLAS_OP_N; break;
		case 1: trans_op = HIPBLAS_OP_T; break;
		case 2: trans_op = HIPBLAS_OP_C; break;
	}
	return trans_op;
}

hipError_t cuda_blas_matrix_multiply(
	const int32_t device_id,
	const int32_t transa_op, const int32_t transb_op,
	const int32_t m, const int32_t n, const int32_t k,
	const float alpha,
	const float* __restrict a,
	const float* __restrict b,
	const float beta,
	float* __restrict c) {
	hipError_t errorCode = hipSetDevice(device_id);
	if (errorCode != hipSuccess) return errorCode;

	hipDeviceProp_t prop;
	errorCode = hipGetDeviceProperties(&prop, device_id);
	if (errorCode != hipSuccess) return errorCode;

	// A(m,k)
	int row_a = m;
	// A(m, k)
	int col_a = k;

	// B(k,n)
	int row_b = k;
	// B(k,n)
	int col_b = n;

	// C(m,n)
	int row_c = m;
	//C(m,n)
	int col_c = n;

	hipblasOperation_t trans_op_a = cuda_blas_determine_op(transa_op);
	hipblasOperation_t trans_op_b = cuda_blas_determine_op(transb_op);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// .NET does not support marshaling nested arrays between C++ and e.g. C#.
	// If you try, you will get the error message, "There is no marshaling support for nested arrays."
	// The solution: this library will only deal with flatten arrays. If you need to use nested arrays,
	// You must create a wrapper that flattens them before passing them to this function and then
	// unflattens them when this function passes a result back.

	float *d_a, *d_b, *d_c;

	size_t size_a = row_a * col_a * sizeof(float);
	size_t size_b = row_b * col_b * sizeof(float);
	size_t size_c = row_c * col_c * sizeof(float);

	// C(m,n) = A(m,k) * B(k,n)
	errorCode = hipMalloc(&d_a, size_a);
	if (errorCode != hipSuccess) return errorCode;
	errorCode = hipMalloc(&d_b, size_b);
	if (errorCode != hipSuccess) return errorCode;
	errorCode = hipMalloc(&d_c, size_c);
	if (errorCode != hipSuccess) return errorCode;

	errorCode = hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
	if (errorCode != hipSuccess) return errorCode;
	errorCode = hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);
	if (errorCode != hipSuccess) return errorCode;

	// C(m,n) = A(m,k) * B(k,n)
	// cuBLAS was written with FORTRAN in mind, which uses a column-major notation for matrices.
	// Native C/C++ code (and, indeed, C#) use row-major notation. This effectively transposes the matrices (but doesn't require any data to be moved around).
	// A simple solution is to swap the matrices.
	// See: http://mccormickml.com/2015/08/29/matrix-multiplication-with-cublas-example/
	// And see: http://peterwittek.com/cublas-matrix-c-style.html
	// Finally: https://gist.github.com/peterwittek/6303527
	hipblasSgemm(
		handle,
		trans_op_a, trans_op_b,
		col_b, row_a, col_a,
		&alpha,
		d_b, col_b,
		d_a, col_a,
		&beta,
		d_c, col_b);

	errorCode = hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);
	if (errorCode != hipSuccess) return errorCode;

	errorCode = hipFree(d_a);
	if (errorCode != hipSuccess) return errorCode;
	errorCode = hipFree(d_b);
	if (errorCode != hipSuccess) return errorCode;
	errorCode = hipFree(d_c);
	if (errorCode != hipSuccess) return errorCode;

	hipblasDestroy(handle);

	return hipSuccess;
}

hipError_t cuda_blas_matrix_multiply(
	const int32_t device_id,
	const int32_t transa_op, const int32_t transb_op,
	const int32_t m, const int32_t n, const int32_t k,
	const double alpha,
	const double* __restrict a,
	const double* __restrict b,
	const double beta,
	double* __restrict c) {
	hipError_t errorCode = hipSetDevice(device_id);
	if (errorCode != hipSuccess) return errorCode;

	hipDeviceProp_t prop;
	errorCode = hipGetDeviceProperties(&prop, device_id);
	if (errorCode != hipSuccess) return errorCode;

	// A(m,k)
	int row_a = m;
	// A(m, k)
	int col_a = k;

	// B(k,n)
	int row_b = k;
	// B(k,n)
	int col_b = n;

	// C(m,n)
	int row_c = m;
	//C(m,n)
	int col_c = n;

	hipblasOperation_t trans_op_a = cuda_blas_determine_op(transa_op);
	hipblasOperation_t trans_op_b = cuda_blas_determine_op(transb_op);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// .NET does not support marshaling nested arrays between C++ and e.g. C#.
	// If you try, you will get the error message, "There is no marshaling support for nested arrays."
	// The solution: this library will only deal with flatten arrays. If you need to use nested arrays,
	// You must create a wrapper that flattens them before passing them to this function and then
	// unflattens them when this function passes a result back.

	double *d_a, *d_b, *d_c;

	size_t size_a = row_a * col_a * sizeof(double);
	size_t size_b = row_b * col_b * sizeof(double);
	size_t size_c = row_c * col_c * sizeof(double);

	// C(m,n) = A(m,k) * B(k,n)
	errorCode = hipMalloc(&d_a, size_a);
	if (errorCode != hipSuccess) return errorCode;
	errorCode = hipMalloc(&d_b, size_b);
	if (errorCode != hipSuccess) return errorCode;
	errorCode = hipMalloc(&d_c, size_c);
	if (errorCode != hipSuccess) return errorCode;

	errorCode = hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
	if (errorCode != hipSuccess) return errorCode;
	errorCode = hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);
	if (errorCode != hipSuccess) return errorCode;

	// C(m,n) = A(m,k) * B(k,n)
	// cuBLAS was written with FORTRAN in mind, which uses a column-major notation for matrices.
	// Native C/C++ code (and, indeed, C#) use row-major notation. This effectively transposes the matrices (but doesn't require any data to be moved around).
	// A simple solution is to swap the matrices.
	// See: http://mccormickml.com/2015/08/29/matrix-multiplication-with-cublas-example/
	// And see: http://peterwittek.com/cublas-matrix-c-style.html
	// Finally: https://gist.github.com/peterwittek/6303527
	hipblasDgemm(
		handle,
		trans_op_a, trans_op_b,
		col_b, row_a, col_a,
		&alpha,
		d_b, col_b,
		d_a, col_a,
		&beta,
		d_c, col_b);

	errorCode = hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);
	if (errorCode != hipSuccess) return errorCode;

	errorCode = hipFree(d_a);
	if (errorCode != hipSuccess) return errorCode;
	errorCode = hipFree(d_b);
	if (errorCode != hipSuccess) return errorCode;
	errorCode = hipFree(d_c);
	if (errorCode != hipSuccess) return errorCode;

	hipblasDestroy(handle);

	return hipSuccess;
}

extern "C" {
	__declspec(dllexport) int MatrixMultiplyFloat(
		int32_t device_id,
		int32_t transa_op, int32_t transb_op,
		int32_t m, int32_t n, int32_t k,
		float alpha,
		float *a,
		float *b,
		float beta,
		float *c) {
		return marshal_cuda_error(cuda_blas_matrix_multiply(device_id, transa_op, transb_op, m, n, k, alpha, a, b, beta, c));
	}

	__declspec(dllexport) int MatrixMultiplyDouble(
		int32_t device_id,
		int32_t transa_op, int32_t transb_op,
		int32_t m, int32_t n, int32_t k,
		double alpha,
		double *a,
		double *b,
		double beta,
		double *c) {
		return marshal_cuda_error(cuda_blas_matrix_multiply(device_id, transa_op, transb_op, m, n, k, alpha, a, b, beta, c));
	}
}
