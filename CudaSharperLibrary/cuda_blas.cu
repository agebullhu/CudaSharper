#include "cuda_blas.h"

hipblasOperation_t cuda_blas_determine_op(unsigned int op) {
	hipblasOperation_t trans_op = HIPBLAS_OP_N;
	switch (op) {
	case 0: trans_op = HIPBLAS_OP_N; break;
	case 1: trans_op = HIPBLAS_OP_T; break;
	case 2: trans_op = HIPBLAS_OP_C; break;
	}
	return trans_op;
}

void cuda_blas_matrix_multiply(
	unsigned int device_id,
	unsigned int transa_op, unsigned int transb_op,
	int m, int n, int k,
	float alpha,
	float *a,
	float *b,
	float beta,
	float *c) {
	hipError_t gpu_device = hipSetDevice(device_id);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device_id);

	// A(m,k)
	int row_a = m;
	// A(m, k)
	int col_a = k;

	// B(k,n)
	int row_b = k;
	// B(k,n)
	int col_b = n;

	// C(m,n)
	int row_c = m;
	//C(m,n)
	int col_c = n;

	hipblasOperation_t trans_op_a = cuda_blas_determine_op(transa_op);
	hipblasOperation_t trans_op_b = cuda_blas_determine_op(transb_op);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// .NET does not support marshaling nested arrays between C++ and e.g. C#.
	// If you try, you will get the error message, "There is no marshaling support for nested arrays."
	// The solution: this library will only deal with flatten arrays. If you need to use nested arrays,
	// You must create a wrapper that flattens them before passing them to this function and then
	// unflattens them when this function passes a result back.

	float *d_a, *d_b, *d_c;

	size_t size_a = row_a * col_a * sizeof(float);
	size_t size_b = row_b * col_b * sizeof(float);
	size_t size_c = row_c * col_c * sizeof(float);

	// C(m,n) = A(m,k) * B(k,n)
	hipMalloc(&d_a, size_a);
	hipMalloc(&d_b, size_b);
	hipMalloc(&d_c, size_c);

	hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);

	// C(m,n) = A(m,k) * B(k,n)
	// cuBLAS was written with FORTRAN in mind, which uses a column-major notation for matrices.
	// Native C/C++ code (and, indeed, C#) use row-major notation. This effectively transposes the matrices (but doesn't require any data to be moved around).
	// A simple solution is to swap the matrices.
	// See: http://mccormickml.com/2015/08/29/matrix-multiplication-with-cublas-example/
	// And see: http://peterwittek.com/cublas-matrix-c-style.html
	// Finally: https://gist.github.com/peterwittek/6303527
	hipblasSgemm(
		handle,
		trans_op_a, trans_op_b,
		col_b, row_a, col_a,
		&alpha,
		d_b, col_b,
		d_a, col_a,
		&beta,
		d_c, col_b);

	hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	hipblasDestroy(handle);
}

void cuda_blas_matrix_multiply(
	unsigned int device_id,
	unsigned int transa_op, unsigned int transb_op,
	int m, int n, int k,
	double alpha,
	double *a,
	double *b,
	double beta,
	double *c) {
	hipError_t gpu_device = hipSetDevice(device_id);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device_id);

	// A(m,k)
	int row_a = m;
	// A(m, k)
	int col_a = k;

	// B(k,n)
	int row_b = k;
	// B(k,n)
	int col_b = n;

	// C(m,n)
	int row_c = m;
	//C(m,n)
	int col_c = n;

	hipblasOperation_t trans_op_a = cuda_blas_determine_op(transa_op);
	hipblasOperation_t trans_op_b = cuda_blas_determine_op(transb_op);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// .NET does not support marshaling nested arrays between C++ and e.g. C#.
	// If you try, you will get the error message, "There is no marshaling support for nested arrays."
	// The solution: this library will only deal with flatten arrays. If you need to use nested arrays,
	// You must create a wrapper that flattens them before passing them to this function and then
	// unflattens them when this function passes a result back.

	double *d_a, *d_b, *d_c;

	size_t size_a = row_a * col_a * sizeof(double);
	size_t size_b = row_b * col_b * sizeof(double);
	size_t size_c = row_c * col_c * sizeof(double);

	// C(m,n) = A(m,k) * B(k,n)
	hipMalloc(&d_a, size_a);
	hipMalloc(&d_b, size_b);
	hipMalloc(&d_c, size_c);

	hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);

	// C(m,n) = A(m,k) * B(k,n)
	// cuBLAS was written with FORTRAN in mind, which uses a column-major notation for matrices.
	// Native C/C++ code (and, indeed, C#) use row-major notation. This effectively transposes the matrices (but doesn't require any data to be moved around).
	// A simple solution is to swap the matrices.
	// See: http://mccormickml.com/2015/08/29/matrix-multiplication-with-cublas-example/
	// And see: http://peterwittek.com/cublas-matrix-c-style.html
	// Finally: https://gist.github.com/peterwittek/6303527
	hipblasDgemm(
		handle,
		trans_op_a, trans_op_b,
		col_b, row_a, col_a,
		&alpha,
		d_b, col_b,
		d_a, col_a,
		&beta,
		d_c, col_b);

	hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	hipblasDestroy(handle);
}

extern "C" __declspec(dllexport) void MatrixMultiplyFloat(
	unsigned int device_id,
	unsigned int transa_op, unsigned int transb_op,
	int m, int n, int k,
	float alpha,
	float *a,
	float *b,
	float beta,
	float *c) {
	cuda_blas_matrix_multiply(device_id, transa_op, transb_op, m, n, k, alpha, a, b, beta, c);
}

extern "C" __declspec(dllexport) void MatrixMultiplyDouble(
	unsigned int device_id,
	unsigned int transa_op, unsigned int transb_op,
	int m, int n, int k,
	double alpha,
	double *a,
	double *b,
	double beta,
	double *c) {
	cuda_blas_matrix_multiply(device_id, transa_op, transb_op, m, n, k, alpha, a, b, beta, c);
}