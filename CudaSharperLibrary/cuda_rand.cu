#include "hip/hip_runtime.h"
#pragma once
#include "cuda_rand.h"

namespace csl {
	
	hipError_t cuda_rand::uniform_rand(const int64_t amount_of_numbers, float* __restrict result) {
		hipError_t errorCode = hipSetDevice(this->device->device_id());
		if (errorCode != hipSuccess) return errorCode;

		// kernel prefers blocks over threads, but does not like only blocks and no threads.
		int32_t threads = CURAND_NUM_OF_THREADS;
		int32_t blocks = 2;
		this->determine_launch_parameters(&blocks, &threads, amount_of_numbers, this->max_blocks, this->max_threads);

		size_t sharedMem = sizeof(hiprandState_t) * threads;
		kernels::cuda_rand::uniform_rand_kernel << <blocks, threads, sharedMem >> > (cuda_rand::time_seed(), this->device->getf32(), amount_of_numbers);

		errorCode = hipMemcpy(result, this->device->getf32(), amount_of_numbers * sizeof(float), hipMemcpyDeviceToHost);
		if (errorCode != hipSuccess) return errorCode;

		return hipSuccess;
	}

	hipError_t cuda_rand::uniform_rand_double(const int64_t amount_of_numbers, double* __restrict result) {
		hipError_t errorCode = hipSetDevice(this->device->device_id());
		if (errorCode != hipSuccess) return errorCode;

		// kernel prefers blocks over threads, but does not like only blocks and no threads.
		int32_t threads = CURAND_NUM_OF_THREADS;
		int32_t blocks = 2;
		this->determine_launch_parameters(&blocks, &threads, amount_of_numbers, this->max_blocks, this->max_threads);

		size_t sharedMem = sizeof(hiprandState_t) * threads;
		kernels::cuda_rand::uniform_rand_double_kernel << <blocks, threads, sharedMem >> > (cuda_rand::time_seed(), this->device->getf64(), amount_of_numbers);

		errorCode = hipMemcpy(result, this->device->getf64(), amount_of_numbers * sizeof(double), hipMemcpyDeviceToHost);
		if (errorCode != hipSuccess) return errorCode;

		return hipSuccess;
	}

	hipError_t cuda_rand::normal_rand(const int64_t amount_of_numbers, float* __restrict result) {
		hipError_t errorCode = hipSetDevice(this->device->device_id());
		if (errorCode != hipSuccess) return errorCode;

		int32_t threads = CURAND_NUM_OF_THREADS;
		int32_t blocks = 2;
		this->determine_launch_parameters(&blocks, &threads, amount_of_numbers, this->max_blocks, this->max_threads);

		size_t sharedMem = sizeof(hiprandState_t) * threads;
		kernels::cuda_rand::normal_rand_kernel << <blocks, threads, sharedMem >> > (cuda_rand::time_seed(), this->device->getf32(), amount_of_numbers);

		errorCode = hipMemcpy(result, this->device->getf32(), sizeof(float) * amount_of_numbers, hipMemcpyDeviceToHost);
		if (errorCode != hipSuccess) return errorCode;

		return hipSuccess;
	}

	hipError_t cuda_rand::normal_rand_double(const int64_t amount_of_numbers, double* __restrict result) {
		hipError_t errorCode = hipSetDevice(this->device->device_id());
		if (errorCode != hipSuccess) return errorCode;

		int32_t threads = CURAND_NUM_OF_THREADS;
		int32_t blocks = 2;
		this->determine_launch_parameters(&blocks, &threads, amount_of_numbers, this->max_blocks, this->max_threads);

		size_t sharedMem = sizeof(hiprandState_t) * threads;
		kernels::cuda_rand::normal_rand_double_kernel << <blocks, threads, sharedMem >> > (cuda_rand::time_seed(), this->device->getf64(), amount_of_numbers);

		errorCode = hipMemcpy(result, this->device->getf64(), sizeof(double) * amount_of_numbers, hipMemcpyDeviceToHost);
		if (errorCode != hipSuccess) return errorCode;

		return hipSuccess;
	}

	hipError_t cuda_rand::log_normal_rand(const int64_t amount_of_numbers, float* __restrict result, float mean, float stddev) {
		hipError_t errorCode = hipSetDevice(this->device->device_id());
		if (errorCode != hipSuccess) return errorCode;

		int32_t threads = CURAND_NUM_OF_THREADS;
		int32_t blocks = 2;
		this->determine_launch_parameters(&blocks, &threads, amount_of_numbers, this->max_blocks, this->max_threads);

		size_t sharedMem = sizeof(hiprandState_t) * threads;
		kernels::cuda_rand::log_normal_rand_kernel << <blocks, threads, sharedMem >> > (cuda_rand::time_seed(), this->device->getf32(), amount_of_numbers, mean, stddev);

		errorCode = hipMemcpy(result, this->device->getf32(), sizeof(float) * amount_of_numbers, hipMemcpyDeviceToHost);
		if (errorCode != hipSuccess) return errorCode;

		return hipSuccess;
	}

	hipError_t cuda_rand::log_normal_rand_double(const int64_t amount_of_numbers, double* __restrict result, double mean, double stddev) {
		hipError_t errorCode = hipSetDevice(this->device->device_id());
		if (errorCode != hipSuccess) return errorCode;

		int32_t threads = CURAND_NUM_OF_THREADS;
		int32_t blocks = 2;
		this->determine_launch_parameters(&blocks, &threads, amount_of_numbers, this->max_blocks, this->max_threads);

		size_t sharedMem = sizeof(hiprandState_t) * threads;
		kernels::cuda_rand::log_normal_rand_double_kernel << <blocks, threads, sharedMem >> > (cuda_rand::time_seed(), this->device->getf64(), amount_of_numbers, mean, stddev);

		errorCode = hipMemcpy(result, this->device->getf64(), sizeof(double) * amount_of_numbers, hipMemcpyDeviceToHost);
		if (errorCode != hipSuccess) return errorCode;

		return hipSuccess;
	}

	hipError_t cuda_rand::poisson_rand(const int64_t amount_of_numbers, int32_t* __restrict result, double lambda) {
		hipError_t errorCode = hipSetDevice(this->device->device_id());
		if (errorCode != hipSuccess) return errorCode;

		int32_t threads = CURAND_NUM_OF_THREADS;
		int32_t blocks = 2;
		this->determine_launch_parameters(&blocks, &threads, amount_of_numbers, this->max_blocks, this->max_threads);

		size_t sharedMem = sizeof(hiprandState_t) * threads;
		kernels::cuda_rand::poisson_rand_kernel << <blocks, threads, sharedMem >> > (cuda_rand::time_seed(), this->device->getu32(), amount_of_numbers, lambda);

		errorCode = hipMemcpy(result, this->device->getu32(), sizeof(float) * amount_of_numbers, hipMemcpyDeviceToHost);
		if (errorCode != hipSuccess) return errorCode;

		return hipSuccess;
	}

	namespace kernels {
		namespace cuda_rand {
			__global__ void uniform_rand_kernel(const int64_t seed, float* __restrict numbers, const int64_t maximum) {
				extern __shared__ hiprandState_t curandStateShared[];

				int xid = blockIdx.x * blockDim.x + threadIdx.x;
				hiprand_init(seed + xid, 0, 0, &curandStateShared[threadIdx.x]);

				for (int i = xid; i < maximum; i += blockDim.x * gridDim.x) {
					numbers[i] = hiprand_uniform(&curandStateShared[threadIdx.x]);
				}
			}

			__global__ void uniform_rand_double_kernel(const int64_t seed, double* __restrict numbers, const int64_t maximum) {
				extern __shared__ hiprandState_t curandStateShared[];

				int xid = blockIdx.x * blockDim.x + threadIdx.x;
				hiprand_init(seed + xid, 0, 0, &curandStateShared[threadIdx.x]);

				for (int i = xid; i < maximum; i += blockDim.x * gridDim.x) {
					numbers[i] = hiprand_uniform_double(&curandStateShared[threadIdx.x]);
				}
			}

			__global__ void normal_rand_kernel(const int64_t seed, float* __restrict numbers, const int64_t maximum) {
				extern __shared__ hiprandState_t curandStateShared[];

				int idx = blockIdx.x * blockDim.x + threadIdx.x;
				int N = maximum / 2;
				hiprand_init(seed + idx, 0, 0, &curandStateShared[threadIdx.x]);

				for (int i = idx; i < N; i += blockDim.x * gridDim.x) {
					// We could use hiprand_normal4, but it requires hiprandStatePhilox4_32_10_t
					// That struct is 64 bytes. Using Pascal, that will lead to a maximum theoretical occupancy of 75%.
					// Using hiprandState_t, which is 48 bytes, we can achieve a occupancy of 100%.
					// This kernel is compute-bound, so achieving higher memory bandwidth over compute will not improve performance.
					reinterpret_cast<float2*>(numbers)[i] = hiprand_normal2(&curandStateShared[threadIdx.x]);
				}

				for (int i = idx + N * 2; i < maximum; i += idx) {
					numbers[i] = hiprand_normal(&curandStateShared[threadIdx.x]);
				}
			}

			__global__ void normal_rand_double_kernel(const int64_t seed, double* __restrict numbers, const int64_t maximum) {
				extern __shared__ hiprandState_t curandStateShared[];

				int idx = blockIdx.x * blockDim.x + threadIdx.x;
				int N = maximum / 2;
				hiprand_init(seed + idx, 0, 0, &curandStateShared[threadIdx.x]);

				for (int i = idx; i < N; i += blockDim.x * gridDim.x) {
					reinterpret_cast<double2*>(numbers)[i] = hiprand_normal2_double(&curandStateShared[threadIdx.x]);
				}

				for (int i = idx + N * 2; i < maximum; i += idx) {
					numbers[i] = hiprand_normal_double(&curandStateShared[threadIdx.x]);
				}
			}

			__global__ void log_normal_rand_kernel(const int64_t seed, float* __restrict numbers, const int64_t maximum, const float mean, const float stddev) {
				extern __shared__ hiprandState_t curandStateShared[];

				int idx = blockIdx.x * blockDim.x + threadIdx.x;
				int N = maximum / 2;
				hiprand_init(seed + idx, 0, 0, &curandStateShared[threadIdx.x]);

				for (int i = idx; i < N; i += blockDim.x * gridDim.x) {
					reinterpret_cast<float2*>(numbers)[i] = hiprand_log_normal2(&curandStateShared[threadIdx.x], mean, stddev);
				}

				for (int i = idx + N * 2; i < maximum; i += idx) {
					numbers[i] = hiprand_log_normal(&curandStateShared[threadIdx.x], mean, stddev);
				}
			}

			__global__ void log_normal_rand_double_kernel(const int64_t seed, double* __restrict numbers, const int64_t maximum, const double mean, const double stddev) {
				extern __shared__ hiprandState_t curandStateShared[];

				int idx = blockIdx.x * blockDim.x + threadIdx.x;
				int N = maximum / 2;
				hiprand_init(seed + idx, 0, 0, &curandStateShared[threadIdx.x]);

				for (int i = idx; i < N; i += blockDim.x * gridDim.x) {
					reinterpret_cast<double2*>(numbers)[i] = hiprand_log_normal2_double(&curandStateShared[threadIdx.x], mean, stddev);
				}

				for (int i = idx + N * 2; i < maximum; i += idx) {
					numbers[i] = hiprand_log_normal_double(&curandStateShared[threadIdx.x], mean, stddev);
				}
			}

			__global__ void poisson_rand_kernel(const int64_t seed, int32_t* __restrict numbers, const int64_t maximum, const double lambda) {
				extern __shared__ hiprandState_t curandStateShared[];

				int xid = blockIdx.x * blockDim.x + threadIdx.x;
				hiprand_init(seed + xid, 0, 0, &curandStateShared[threadIdx.x]);

				for (int i = xid; i < maximum; i += blockDim.x * gridDim.x) {
					numbers[i] = hiprand_poisson(&curandStateShared[threadIdx.x], lambda);
				}
			}
		}
	}
}
