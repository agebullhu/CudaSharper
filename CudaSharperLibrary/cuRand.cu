#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

__global__ void init(unsigned int seed, hiprandState_t* states) {
	int xid = (threadIdx.x + (blockIdx.x * blockDim.x));
	hiprand_init(
		xid + seed,
		0,
		0,
		&states[xid]
	);
}

__global__ void uniform_rand_kernel(hiprandState_t *states, float *numbers, int count) {
	int xid = (threadIdx.x + (blockIdx.x * blockDim.x));

	// This is an offset: it determines the starting point of this kernel's place in the array.
	int offset = (threadIdx.x + (blockIdx.x * blockDim.x)) * count;

	for (int n = offset; n < offset + count; n++) {
		numbers[n] = hiprand_uniform(&states[xid]);
	}
}

void _uniformRand(int device_id, int amount_of_numbers, float *result) {
	hipError_t gpu_device = hipSetDevice(device_id);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device_id);

	// uniform_rand_kernel prefers blocks over threads, but does not like only blocks and no threads.
	const int threadsPerBlock = prop.warpSize * 2;
	const int blocks = *prop.maxThreadsDim;
	const int numberPerThread = (amount_of_numbers / (blocks * threadsPerBlock)) + 1;

	hiprandState_t *states;
	float *d_nums;

	hipMalloc(&states, blocks * threadsPerBlock * sizeof(hiprandState_t));
	hipMalloc(&d_nums, amount_of_numbers * sizeof(float));

	init << <blocks, threadsPerBlock >> > (time(0), states);
	uniform_rand_kernel << <blocks, threadsPerBlock >> > (states, d_nums, numberPerThread);

	hipMemcpy(result, d_nums, amount_of_numbers * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(states);
	hipFree(d_nums);
}

extern "C" __declspec(dllexport) void UniformRand(int device_id, int amount_of_numbers, float *result) {
	_uniformRand(device_id, amount_of_numbers, result);
}

__global__ void normal_rand_kernel(hiprandState_t *states, float *numbers, int count) {
	int xid = (threadIdx.x + (blockIdx.x * blockDim.x));

	// This is an offset: it determines the starting point of this kernel's place in the array.
	int offset = (threadIdx.x + (blockIdx.x * blockDim.x)) * count;

	for (int n = offset; n < offset + count; n++) {
		numbers[n] = hiprand_normal(&states[xid]);
	}
}

void _normalRand(int device_id, int amount_of_numbers, float *result) {
	hipError_t gpu_device = hipSetDevice(device_id);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device_id);

	// uniform_rand_kernel prefers blocks over threads, but does not like only blocks and no threads.
	const int threadsPerBlock = prop.warpSize * 2;
	const int blocks = *prop.maxThreadsDim;
	const int numberPerThread = (amount_of_numbers / (blocks * threadsPerBlock)) + 1;

	hiprandState_t *states;
	float *d_nums;

	hipMalloc(&states, blocks * threadsPerBlock * sizeof(hiprandState_t));
	hipMalloc(&d_nums, amount_of_numbers * sizeof(float));

	init << <blocks, threadsPerBlock >> > (time(0), states);
	normal_rand_kernel << <blocks, threadsPerBlock >> > (states, d_nums, numberPerThread);

	hipMemcpy(result, d_nums, amount_of_numbers * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(states);
	hipFree(d_nums);
}

extern "C" __declspec(dllexport) void NormalRand(int device_id, int amount_of_numbers, float *result) {
	_normalRand(device_id, amount_of_numbers, result);
}

__global__ void poisson_rand_kernel(hiprandState_t *states, int *numbers, double lambda, int count) {
	int xid = (threadIdx.x + (blockIdx.x * blockDim.x));

	// This is an offset: it determines the starting point of this kernel's place in the array.
	int offset = (threadIdx.x + (blockIdx.x * blockDim.x)) * count;

	for (int n = offset; n < offset + count; n++) {
		numbers[n] = hiprand_poisson(&states[xid], lambda);
	}
}

void _poissonRand(int device_id, int amount_of_numbers, int *result, double lambda) {
	hipError_t gpu_device = hipSetDevice(device_id);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device_id);

	// uniform_rand_kernel prefers blocks over threads, but does not like only blocks and no threads.
	const int threadsPerBlock = prop.warpSize * 2;
	const int blocks = *prop.maxThreadsDim;
	const int numberPerThread = (amount_of_numbers / (blocks * threadsPerBlock)) + 1;

	hiprandState_t *states;
	int *d_nums;

	hipMalloc(&states, blocks * threadsPerBlock * sizeof(hiprandState_t));
	hipMalloc(&d_nums, amount_of_numbers * sizeof(int));

	init << <blocks, threadsPerBlock >> > (time(0), states);
	poisson_rand_kernel << <blocks, threadsPerBlock >> > (states, d_nums, lambda, numberPerThread);

	hipMemcpy(result, d_nums, amount_of_numbers * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(states);
	hipFree(d_nums);
}

extern "C" __declspec(dllexport) void PoissonRand(int device_id, int amount_of_numbers, int *result, double lambda) {
	_poissonRand(device_id, amount_of_numbers, result, lambda);
}
