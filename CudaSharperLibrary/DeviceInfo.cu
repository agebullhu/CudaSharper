#include "DeviceInfo.h"

int32_t marshal_cuda_error(hipError_t error) {
	return (int)error;
}

std::atomic<bool>* DeviceInfo::is_context_initialized = new std::atomic<bool>(false);
std::atomic<bool>* DeviceInfo::is_device_prop_initialized;
hipDeviceProp_t* DeviceInfo::properties;

DeviceInfo::DeviceInfo(int32_t device_id) {
	if (!DeviceInfo::is_context_initialized->load()) {
		int32_t num = this->get_cuda_device_count();
		if (num > 0) {
			DeviceInfo::is_device_prop_initialized = (std::atomic<bool>*)malloc(sizeof(std::atomic<bool>) * num);
			DeviceInfo::properties = (hipDeviceProp_t*)malloc(sizeof(hipDeviceProp_t) * num);

			// This loads in the CUDA context and reduces the time needed to do things like hipMalloc (subsequent calls will be faster regardless).
			for (int i = 0; i < num; i++) {
				hipSetDevice(i);
				hipFree(0);
				this->get_device_properties(i, &properties[i]);
			}

			DeviceInfo::is_context_initialized->store(true);
		}
	}
	this->device_id = device_id;
}

hipError_t DeviceInfo::get_device_properties(int32_t device_id, hipDeviceProp_t *prop) const {
	/*if (DeviceInfo::is_device_prop_initialized[device_id].load() != true) {
		hipError_t errorCode = hipGetDeviceProperties(&DeviceInfo::properties[device_id], device_id);
		if (errorCode != hipSuccess) return errorCode;
		DeviceInfo::is_device_prop_initialized[device_id].store(true);
	}

	*prop = DeviceInfo::properties[device_id];*/

	hipGetDeviceProperties(prop, device_id);

	return hipSuccess;
}

int32_t DeviceInfo::get_cuda_device_count() const {
	int32_t num = 0;
	this->get_cuda_device_count(num);
	return num;
}

hipError_t DeviceInfo::get_cuda_device_count(int32_t& result) const {
	int num = 0;
	hipError_t errorCode = hipGetDeviceCount(&num);
	if (errorCode != hipSuccess) return errorCode;

	result = num;

	return hipSuccess;
}

hipError_t DeviceInfo::get_cuda_device_name(int32_t device_id, char* device_name_ptr) const {
	hipDeviceProp_t prop;
	// The following isn't working correctly. Eventually, it should be used, because then we can save time calling hipGetDeviceProperties
	hipError_t errorCode = this->get_device_properties(device_id, &prop);
	if (errorCode != hipSuccess) return errorCode;

	// Length of hipDeviceProp_t::name, according to current NVIDIA documentation: http://docs.nvidia.com/cuda/cuda-runtime-api/structcudaDeviceProp.html#structcudaDeviceProp_11e26f1c6bd42f4821b7ef1a4bd3bd25c
	strncpy(device_name_ptr, prop.name, 256);

	return hipSuccess;
}

hipError_t DeviceInfo::reset_cuda_device() const {
	hipError_t errorCode;
	for (int i = 0; i < this->get_cuda_device_count(); i++) {
		errorCode = hipSetDevice(i);
		if (errorCode != hipSuccess) return errorCode;
		errorCode = hipDeviceReset();
		if (errorCode != hipSuccess) return errorCode;
	}
	return hipSuccess;
}

int32_t DeviceInfo::get_device_id() const {
	return this->device_id;
}

extern "C" {
	__declspec(dllexport) int32_t GetCudaDeviceCount() {
		DeviceInfo device;
		int32_t num = 0;
		device.get_cuda_device_count(num);
		return num;
	}

	__declspec(dllexport) int32_t GetCudaDeviceName(int32_t device_id, char* device_name_ptr) {
		DeviceInfo device;
		return marshal_cuda_error(device.get_cuda_device_name(device_id, device_name_ptr));
	}

	__declspec(dllexport) int32_t ResetCudaDevice() {
		DeviceInfo device;
		return marshal_cuda_error(device.reset_cuda_device());
	}
}