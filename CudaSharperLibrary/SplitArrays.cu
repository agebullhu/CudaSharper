#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

__global__ void copyArrays(int *result, int *input, const unsigned int offset, const unsigned int length)
{
	int xid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (xid < length) {
		result[xid] = input[xid + offset];
	}
}

template<typename T> void splitArray(int device_id, int *src, int *array1, int *array2, const unsigned int array_length, const unsigned int split_index)
{
	hipError_t gpu_device = hipSetDevice(device_id);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device_id);

	const int threadsPerBlock = prop.maxThreadsPerBlock;
	const int blocks = (array_length / threadsPerBlock) + 1;

	const int array1_length = split_index, array2_length = array_length - split_index;

	T *d_src, *d_array1, *d_array2;

	hipMalloc(&d_src, array_length * sizeof(T));
	hipMalloc(&d_array1, array1_length * sizeof(T));
	hipMalloc(&d_array2, array2_length * sizeof(T));

	hipStream_t stream1;
	hipStreamCreate(&stream1);

	hipMemcpyAsync(d_src, src, array_length * sizeof(T), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(d_array1, array1, array1_length * sizeof(T), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(d_array2, array2, array2_length * sizeof(T), hipMemcpyHostToDevice, stream1);

	hipStreamSynchronize(stream1);

	hipStream_t stream2;
	hipStreamCreate(&stream2);

	copyArrays << <blocks, threadsPerBlock, 0, stream1 >> > (d_array1, d_src, 0, array1_length);
	copyArrays << <blocks, threadsPerBlock, 0, stream2 >> > (d_array2, d_src, split_index, array2_length);

	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);

	hipMemcpyAsync(array1, d_array1, array1_length * sizeof(T), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(array2, d_array2, array2_length * sizeof(T), hipMemcpyDeviceToHost, stream1);

	hipStreamSynchronize(stream1);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);

	hipFree(d_array1);
	hipFree(d_array2);
}

extern "C" __declspec(dllexport) void SplitIntArray(int device_id, int *src, int *array1, int *array2, const unsigned int array_length, const unsigned int split_index);

void SplitIntArray(int device_id, int *src, int *array1, int *array2, const unsigned int array_length, const unsigned int split_index) {
	splitArray<int>(device_id, src, array1, array2, array_length, split_index);
}
